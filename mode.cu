#include "hip/hip_runtime.h"
//--コンパイル方法--
// nvcc mode.cu
// 以下のバージョンでコンパイルができることを確認
// nvcc: NVIDIA (R) Cuda compiler driver
// Copyright (c) 2005-2020 NVIDIA Corporation
// Built on Tue_Sep_15_19:10:02_PDT_2020
// Cuda compilation tools, release 11.1, V11.1.74
// Build cuda_11.1.TC455_06.29069683_0


#include <iostream>
#include <vector>
#include <unordered_map>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/unique.h>
using namespace std;

#define MAXSEED 10 // increase this if you want to take more time data measurings.
#define INTMAX 9999
#define WARPSIZE 32
#define LOG2WARP 5 //log2(32)
#define WARP_BLOCK 4 //warps per block
#define THREAD_BLOCK 128 //WARPSIZE * WARP_BLOCK
#define NUMS_WARP 256 // the number of data checked by a warp
#define NUMS_BLOCK 1024 //NUMS_WARP * WARP_BLOCK
#define MODEMINCNT 16 // N / M (same as minPossibleCnt in main)


void errorCheck(hipError_t ret);
__device__ int nextUniqueIdx(int localtid, int localwid, uint64_t *nums, int *tmp, int baseidx, int N);
__global__ void countingKernel(uint64_t *nums, int N, uint64_t *uniqueNums, int *numsCnt);
void generate_testcase(uint64_t* a, int seed, int N, int M);
uint64_t calcModeHost(uint64_t *a, int N);


int main() {
    uint64_t hostAns = 0; // mode calculated by CPU
    uint64_t deviceAns = 0; // calculated by GPU
    const int N = 1 << 24;
    const int M = 1 << 20;
    const int minPossibleCnt = N / M; // the mode's counts are at least 16 (N / M).
    uint64_t* nums_h = (uint64_t*)malloc(sizeof(uint64_t) * N);
    uint64_t* nums_d;
    uint64_t* uniqueNums_d; // keeps unique numbers which appears more than 16(minPossibleCnt) times.
    int* numsCnt_d; // keeps the counts of uniqueNums_d.

    float measuredTimes[3];
    float averagedTimes[3] = {0, 0, 0};

    errorCheck(hipMalloc((void**)&nums_d, sizeof(uint64_t) * N));
    errorCheck(hipMalloc((void**)&uniqueNums_d, sizeof(uint64_t) * N / minPossibleCnt)); // since only counts nums appearing 16(minPossibleCnt) times, N / minPossibleCnt is enough.
    errorCheck(hipMalloc((void**)&numsCnt_d, sizeof(int) * N / minPossibleCnt));

    thrust::device_ptr<uint64_t> numsDevptr = thrust::device_pointer_cast(nums_d); //ptr for thrust
    thrust::device_ptr<int> numsCntDevptr = thrust::device_pointer_cast(numsCnt_d); //ptr for thrust
    thrust::device_ptr<uint64_t> uniqueNumsDevptr = thrust::device_pointer_cast(uniqueNums_d); //ptr for thrust

    // time recorders
    hipEvent_t start1, stop1, start2, stop2, start3, stop3;
    errorCheck(hipEventCreate(&start1));
    errorCheck(hipEventCreate(&stop1));
    errorCheck(hipEventCreate(&start2));
    errorCheck(hipEventCreate(&stop2));
    errorCheck(hipEventCreate(&start3));
    errorCheck(hipEventCreate(&stop3));

    for (int seed = 0; seed <= MAXSEED; seed++) {
        // generate inputs
        generate_testcase(nums_h, seed, N, M);

        // calculate mode on CPU
        hostAns = calcModeHost(nums_h, N);

        // transfer inputs to device
        errorCheck(hipMemcpy(nums_d, nums_h, sizeof(uint64_t) * N, hipMemcpyHostToDevice));

        //phase1 : sorting
        hipEventRecord(start1);

        thrust::sort(numsDevptr, numsDevptr + N);

        errorCheck(hipEventRecord(stop1));
        errorCheck(hipEventSynchronize(stop1));
        errorCheck(hipEventElapsedTime(&measuredTimes[0], start1, stop1));

        // phase2 : counting unique numbers
        errorCheck(hipEventRecord(start2));
        countingKernel<<< N / NUMS_BLOCK, THREAD_BLOCK >>>(nums_d, N, uniqueNums_d, numsCnt_d);

        errorCheck(hipEventRecord(stop2));
        errorCheck(hipEventSynchronize(stop2));
        errorCheck(hipEventElapsedTime(&measuredTimes[1], start2, stop2));

        // phase3 : find mode by max-reduction of unique numbers' count. (more strictly, find offset where the mode is in uniqueNums_d)
        errorCheck(hipEventRecord(start3));
        thrust::device_ptr<int> modeCntDevptr = thrust::max_element(numsCntDevptr, numsCntDevptr + N / minPossibleCnt);
        // thrust::max_element returns the "first" iterator i which points the largest value (from thrust documentation)
        // so this will return the smallest mode.
        errorCheck(hipEventRecord(stop3));
        errorCheck(hipEventSynchronize(stop3));
        errorCheck(hipEventElapsedTime(&measuredTimes[2], start3, stop3));

        int modeIdx = modeCntDevptr - numsCntDevptr; // offset of the mode's position in uniqueNums_d (and numsCnt_d)

        errorCheck(hipMemcpy(&deviceAns, uniqueNums_d + modeIdx, sizeof(uint64_t), hipMemcpyDeviceToHost));
        
        if (hostAns != deviceAns) { // verify the calculated mode
            cout << "host answer and device answer doesn't match in seed " << seed << "!" << endl;
            cout << "host answer :" << hostAns << endl;
            cout << "device answer :" << deviceAns << endl;
        }

        if (seed != 0) { // discard fist data considering the warming up of GPU.
            averagedTimes[0] += measuredTimes[0];
            averagedTimes[1] += measuredTimes[1];
            averagedTimes[2] += measuredTimes[2];
        }
    }

    averagedTimes[0] /= MAXSEED;
    averagedTimes[1] /= MAXSEED;
    averagedTimes[2] /= MAXSEED;

    cout << "// average execution time of " << MAXSEED << " seeds //" << endl;
    cout << "phase1 (sorting) : " << averagedTimes[0] << " msec." << endl;
    cout << "phase2 (counting) : " << averagedTimes[1] << " msec." << endl;
    cout << "phase3 (reduction) : " << averagedTimes[2] << " msec." << endl;
    cout << "total : " << averagedTimes[0] + averagedTimes[1] + averagedTimes[2] << " msec." << endl;


    errorCheck(hipEventDestroy(start1));
    errorCheck(hipEventDestroy(stop1));
    errorCheck(hipEventDestroy(start2));
    errorCheck(hipEventDestroy(stop2));
    errorCheck(hipEventDestroy(start3));
    errorCheck(hipEventDestroy(stop3));


    return 0;
}

void errorCheck(hipError_t ret) {
    if (ret != hipSuccess) {
        printf("CUDA Error:%s\n", hipGetErrorString(ret));
        exit(-1);
    }
}

// calculates the next unique number's idx in a sequential(sorted) numbers.
__device__ int nextUniqueIdx(int localtid, int localwid, uint64_t *nums, int *tmp, int baseidx, int N) {
    if (baseidx + localtid + 1 >= N) { // boundary check
        tmp[threadIdx.x] = localtid + 1;
    }
    else if (nums[baseidx] == nums[baseidx + localtid + 1]) { 
        tmp[threadIdx.x] = INTMAX;
    }
    else {
        tmp[threadIdx.x] = localtid + 1;
    }

    int activeThreads = WARPSIZE / 2;

    for (int i = 0; i < LOG2WARP; i++) { // find min in tmp, which is the localtid of next unique number holder.
        if (localtid < activeThreads) {
            if (tmp[threadIdx.x] > tmp[threadIdx.x+activeThreads]) {
                tmp[threadIdx.x] = tmp[threadIdx.x+activeThreads];
            }
        }
        activeThreads /= 2;
    }
    
    return tmp[localwid * WARPSIZE];
}

__global__ void countingKernel(uint64_t *nums, int N, uint64_t *uniqueNums, int *numsCnt){
    __shared__ int tmp[THREAD_BLOCK]; //working table

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int wid = tid / WARPSIZE;
    int localtid = threadIdx.x % WARPSIZE;
    int localwid = threadIdx.x / WARPSIZE;
    int originBaseidx = wid * NUMS_WARP;
    int baseidx = wid * NUMS_WARP;

    int cntIdx = wid * (NUMS_WARP / MODEMINCNT); //current idx of uniqueNums & numsCnt

    if (localtid / (NUMS_WARP / MODEMINCNT) == 0) {
        uniqueNums[cntIdx + localtid] = 0;
        numsCnt[cntIdx + localtid] = 0;
    }

    if (baseidx != 0 && nums[baseidx] == nums[baseidx-1]) { // if the previous num == start num, then  the previous warp is reponsible for this area
        while(1) { // move baseidx until next unique number appears.
            if (baseidx >= N) {
                return;
            }
            int nextUniqIdx = nextUniqueIdx(localtid, localwid, nums, tmp, baseidx, N);
            if (nextUniqIdx == INTMAX) { // when all 32(warpsize) numbers are same
                baseidx += WARPSIZE;
            }
            else { 
                baseidx += nextUniqIdx;
                break;
            }
        }
    }

    bool carryFlag = false; // when the unique number has more than 32, the flag rise and keep on counting.

    while (baseidx < originBaseidx + NUMS_WARP || carryFlag) {
        int nextUniqIdx = nextUniqueIdx(localtid, localwid, nums, tmp, baseidx, N);
        if (nextUniqIdx == INTMAX) { // when all 32(warpsize) numbers are same
            if (localtid == 0) {
                uniqueNums[cntIdx] = nums[baseidx];
                numsCnt[cntIdx] += WARPSIZE;
            }
            baseidx += WARPSIZE;
            carryFlag = true;
        }
        else { 
            if (!carryFlag && nextUniqIdx < MODEMINCNT) { // when the number count is under 16, it cannot be the mode.
                baseidx += nextUniqIdx;
                carryFlag = false;
            }
            else {
                if (localtid == 0) {
                    uniqueNums[cntIdx] = nums[baseidx];
                    numsCnt[cntIdx] += nextUniqIdx;
                }
                cntIdx++;
                baseidx += nextUniqIdx;
                carryFlag = false;
            }
        }
    }
}

void generate_testcase(uint64_t* a, int seed, int N, int M) {
    mt19937 mt(seed);
    uniform_int_distribution<uint64_t> dist1;
    vector<uint64_t> x(M);
    for (int i = 0; i < M; i++) {
        x[i] = dist1(mt);
    }
    uniform_int_distribution<int> dist2(0, M-1);
    for (size_t i = 0; i < N; i++) {
        a[i] = x[dist2(mt)];
    }
}

uint64_t calcModeHost(uint64_t *a, int N) {
    unordered_map<uint64_t, int> numcnt; 
    for (int i = 0; i < N; i++) {
        if (numcnt.find(a[i]) == numcnt.end()) {
            numcnt.emplace(a[i], 1);
        }
        else {
            numcnt.at(a[i])++;
        }
    }
    uint64_t mode = 0;
    int modeCnt = 0;
    for (auto itr = numcnt.begin(); itr != numcnt.end(); itr++) {
        if (modeCnt == itr->second && mode > itr->first) {
            mode = itr->first;
            modeCnt = itr->second;
        }
        else if (modeCnt < itr->second) {
            mode = itr->first;
            modeCnt = itr->second;
        }
    }
    return mode;
}